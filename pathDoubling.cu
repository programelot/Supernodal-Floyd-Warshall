#include "hip/hip_runtime.h"
#include <stdio.h>
#include <papi.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
    if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
    file, line );
    }
}

__device__ unsigned int decomposeCUDA(int nVertex, int i, int j = 0){
    return i + j * nVertex;
}

__global__ void updateEdge(int threadSize, int nVertex, unsigned int* apsp){
    int idx = threadIdx.x + blockIdx.x * threadSize;
    if(idx > nVertex * nVertex)
        return;
    int i = idx % nVertex; 
    int j = (idx / nVertex) % nVertex;
    for(int k = 0; k < nVertex; ++k){
        if(apsp[decomposeCUDA(nVertex,i,j)] > apsp[decomposeCUDA(nVertex,i,k)] + apsp[decomposeCUDA(nVertex,k,j)]){
            apsp[decomposeCUDA(nVertex,i,j)] = apsp[decomposeCUDA(nVertex,i,k)] + apsp[decomposeCUDA(nVertex,k,j)];
        }
    }
}

unsigned int decomposeGPU(int nVertex, int i, int j = 0){
    return i + j * nVertex;
}

void pathDouble(unsigned int nVertex, unsigned int* apsp){
    unsigned long long allFrom = PAPI_get_real_usec();
    unsigned int* apsp_cuda;

    HANDLE_ERROR(hipMalloc((void**)&apsp_cuda, sizeof(unsigned int) * nVertex * nVertex));
    
    HANDLE_ERROR(hipMemcpy(apsp_cuda, apsp, sizeof(unsigned int) * nVertex * nVertex, hipMemcpyHostToDevice));
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    unsigned long long computingSize = nVertex * nVertex ;
    unsigned long long threadSize = prop.maxThreadsPerBlock;
    unsigned long long blockSize = (computingSize + threadSize - 1)/threadSize;
    if(threadSize > computingSize){
        threadSize = computingSize;
    }
    int time = 0;
    int currentDist = 1;
    while(currentDist < nVertex - 1){
        ++time;
        currentDist *= 2;
    }
    unsigned long long allTo = PAPI_get_real_usec();
    for(int i = 0; i < time; ++i)
        updateEdge<<<blockSize, threadSize>>>(threadSize, nVertex, apsp_cuda);

    unsigned long long freeFrom = PAPI_get_real_usec();
    HANDLE_ERROR(hipMemcpy(apsp, apsp_cuda, sizeof(unsigned int) * nVertex * nVertex, hipMemcpyDeviceToHost));
    
    HANDLE_ERROR(hipFree(apsp_cuda));
    
    unsigned long long freeTo = PAPI_get_real_usec();
    printf("%d %d %d %d %lu %lu\n", threadSize, blockSize, computingSize, time, allTo - allFrom, freeTo - freeFrom);
}